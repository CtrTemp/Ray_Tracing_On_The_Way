#include "hip/hip_runtime.h"
#include "render.h"
#define CUDA_LAUNCH_BLOCKING

// 写图像文件
__host__ static void write_file(std::string file_path, vec3 *frame_buffer);

__host__ static void showFrameFlow(int width, int height, vec3 *frame_buffer_host);

/* #################################### 纹理贴图初始化 #################################### */
__host__ static void import_tex()
{
    std::string test_texture_path;
    uchar4 *texture_host;
    int texWidth;
    int texHeight;
    int texChannels;
    int texSize;
    size_t pixel_num;

    /* ##################################### Skybox-Front ##################################### */
    test_texture_path = "../Pic/skybox_sunset/Sky_FantasySky_Fire_Cam_0_Front+Z.png";
    texture_host = load_image_texture_host(test_texture_path, &texWidth, &texHeight, &texChannels);
    texSize = texWidth * texHeight * texChannels;
    pixel_num = texWidth * texHeight;

    hipArray *cuArray_skybox_front;                                                        // CUDA 数组类型定义
    hipChannelFormatDesc channelDesc_skybox_front = hipCreateChannelDesc<uchar4>();       // 这一步是建立映射？？
    hipMallocArray(&cuArray_skybox_front, &channelDesc_skybox_front, texWidth, texHeight); // 为array申请显存空间
    hipBindTextureToArray(texRef2D_SkyBox_Front, cuArray_skybox_front);
    hipMemcpyToArray(cuArray_skybox_front, 0, 0, texture_host, sizeof(uchar4) * texWidth * texHeight, hipMemcpyHostToDevice);

    /* ##################################### Skybox-Back ##################################### */
    test_texture_path = "../Pic/skybox_sunset/Sky_FantasySky_Fire_Cam_1_Back-Z.png";
    texture_host = load_image_texture_host(test_texture_path, &texWidth, &texHeight, &texChannels);
    texSize = texWidth * texHeight * texChannels;
    pixel_num = texWidth * texHeight;

    hipArray *cuArray_skybox_back;                                                       // CUDA 数组类型定义
    hipChannelFormatDesc channelDesc_skybox_back = hipCreateChannelDesc<uchar4>();      // 这一步是建立映射？？
    hipMallocArray(&cuArray_skybox_back, &channelDesc_skybox_back, texWidth, texHeight); // 为array申请显存空间
    hipBindTextureToArray(texRef2D_SkyBox_Back, cuArray_skybox_back);
    hipMemcpyToArray(cuArray_skybox_back, 0, 0, texture_host, sizeof(uchar4) * texWidth * texHeight, hipMemcpyHostToDevice);

    /* ##################################### Skybox-Left ##################################### */
    test_texture_path = "../Pic/skybox_sunset/Sky_FantasySky_Fire_Cam_2_Left+X.png";
    texture_host = load_image_texture_host(test_texture_path, &texWidth, &texHeight, &texChannels);
    texSize = texWidth * texHeight * texChannels;
    pixel_num = texWidth * texHeight;

    hipArray *cuArray_skybox_left;                                                       // CUDA 数组类型定义
    hipChannelFormatDesc channelDesc_skybox_left = hipCreateChannelDesc<uchar4>();      // 这一步是建立映射？？
    hipMallocArray(&cuArray_skybox_left, &channelDesc_skybox_left, texWidth, texHeight); // 为array申请显存空间
    hipBindTextureToArray(texRef2D_SkyBox_Left, cuArray_skybox_left);
    hipMemcpyToArray(cuArray_skybox_left, 0, 0, texture_host, sizeof(uchar4) * texWidth * texHeight, hipMemcpyHostToDevice);

    /* ##################################### Skybox-Right ##################################### */
    test_texture_path = "../Pic/skybox_sunset/Sky_FantasySky_Fire_Cam_3_Right-X.png";
    texture_host = load_image_texture_host(test_texture_path, &texWidth, &texHeight, &texChannels);
    texSize = texWidth * texHeight * texChannels;
    pixel_num = texWidth * texHeight;

    hipArray *cuArray_skybox_right;                                                        // CUDA 数组类型定义
    hipChannelFormatDesc channelDesc_skybox_right = hipCreateChannelDesc<uchar4>();       // 这一步是建立映射？？
    hipMallocArray(&cuArray_skybox_right, &channelDesc_skybox_right, texWidth, texHeight); // 为array申请显存空间
    hipBindTextureToArray(texRef2D_SkyBox_Right, cuArray_skybox_right);
    hipMemcpyToArray(cuArray_skybox_right, 0, 0, texture_host, sizeof(uchar4) * texWidth * texHeight, hipMemcpyHostToDevice);

    /* ##################################### Skybox-Up ##################################### */
    test_texture_path = "../Pic/skybox_sunset/Sky_FantasySky_Fire_Cam_4_Up+Y.png";
    texture_host = load_image_texture_host(test_texture_path, &texWidth, &texHeight, &texChannels);
    texSize = texWidth * texHeight * texChannels;
    pixel_num = texWidth * texHeight;

    hipArray *cuArray_skybox_up;                                                     // CUDA 数组类型定义
    hipChannelFormatDesc channelDesc_skybox_up = hipCreateChannelDesc<uchar4>();    // 这一步是建立映射？？
    hipMallocArray(&cuArray_skybox_up, &channelDesc_skybox_up, texWidth, texHeight); // 为array申请显存空间
    hipBindTextureToArray(texRef2D_SkyBox_Up, cuArray_skybox_up);
    hipMemcpyToArray(cuArray_skybox_up, 0, 0, texture_host, sizeof(uchar4) * texWidth * texHeight, hipMemcpyHostToDevice);

    /* ##################################### Skybox-Down ##################################### */
    test_texture_path = "../Pic/skybox_sunset/Sky_FantasySky_Fire_Cam_5_Down-Y.png";
    texture_host = load_image_texture_host(test_texture_path, &texWidth, &texHeight, &texChannels);
    texSize = texWidth * texHeight * texChannels;
    pixel_num = texWidth * texHeight;

    hipArray *cuArray_skybox_down;                                                       // CUDA 数组类型定义
    hipChannelFormatDesc channelDesc_skybox_down = hipCreateChannelDesc<uchar4>();      // 这一步是建立映射？？
    hipMallocArray(&cuArray_skybox_down, &channelDesc_skybox_down, texWidth, texHeight); // 为array申请显存空间
    hipBindTextureToArray(texRef2D_SkyBox_Down, cuArray_skybox_down);
    hipMemcpyToArray(cuArray_skybox_down, 0, 0, texture_host, sizeof(uchar4) * texWidth * texHeight, hipMemcpyHostToDevice);
}

/* ##################################### 随机数初始化 ##################################### */

__global__ void initialize_device_random(hiprandStateXORWOW *states, unsigned long long seed, size_t size)
{
    int row_index = blockDim.y * blockIdx.y + threadIdx.y; // 当前线程所在行索引
    int col_index = blockDim.x * blockIdx.x + threadIdx.x; // 当前线程所在列索引
    if ((row_index >= FRAME_HEIGHT) || (col_index >= FRAME_WIDTH))
    {
        return;
    }
    int row_len = FRAME_WIDTH; // 行宽（列数）
    // int col_len = FRAME_HEIGHT;                 // 列高（行数）
    int global_index = (row_len * row_index + col_index); // 全局索引

    // int *i = new int;
    // *i = 0;
    // printf("i=%d,", *i);
    // delete i; // 这里如果不delete的话会导致显存溢出的错误？？
    // // 这里应该得到启发，之前的一些错误可能是内存/显存栈溢出造成的
    hiprand_init(seed, global_index, 0, &states[global_index]);
}

/* ##################################### 摄像机初始化 ##################################### */

/* ##################################### 场景初始化 ##################################### */
// 最后两个参数是需要创建的 models，需要时，应该在host端预先对其进行初始化，并在device端进行空间分配/拷贝
__global__ void gen_world(hiprandStateXORWOW *rand_state, hitable_list **world, hitable **list, vertex *vertList, uint32_t *indList, int *vertOffset, int *indOffset, int model_counts)
{

    // // 使用 new 关键字相当于在全局内存上开辟空间并创建变量
    // dielectric *glass_aa = new dielectric(1.5);
    // lambertian *aa = new lambertian(new noise_texture(2.5, rand_state));
    // // 但这种创建极其耗费时间，故无法使用以下的函数？？？
    // // 以上的创建只是指明了开辟了一个地址，但似乎并没有为对象实例开辟空间，所以下列的类成员函数根本访问不到？？
    // // 真的是这个原因么
    // printf("aa has emission?? %d\n", aa->hasEmission(0));

    // float aa_pdf = aa->pdf(vec3(1, 2, 3), vec3(1, 2, 3), vec3(1, 2, 3));
    // // 但奇怪的是这个pdf计算函数却可以访问的到
    // printf("aa_pdf = %f \n", aa_pdf);
    // // 让我们再来尝试访问一下其他的类成员变量，同样可以访问
    // printf("aa self tpye = %d\n", glass_aa->self_type);
    // printf("aa_glass self type = %d\n", glass_aa->self_type);
    // 尝试使用类成员函数访问其类内变量，发现访问失败，说明无法调用这个函数
    // printf("glass_aa self type, %d\n",glass_aa->getMaterialType());

    // 经过对比发现了一个很大的bug，不清楚是否为CUDA内部的bug或者一种规范要求，即：
    // 类内函数应该必须至少传入一个参数，即使我们不使用它，否则它就无法被程序正常寻址，导致错误
    // 于是我给 hasEmission() 函数加了一个空传参数后就可以访问对应的函数了。。。

    // 在设备端创建
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        // 一般表面材质/纹理
        material *noise = new lambertian(new noise_texture(20, rand_state));
        material *diffuse_steelblue = new lambertian(new constant_texture(vec3(0.1, 0.2, 0.5)));
        material *mental_copper = new mental(vec3(0.8, 0.6, 0.2), 0.001);
        material *mental_steel = new mental(vec3(0.99, 0.99, 0.99), 0.001);
        material *mental_ground = new mental(vec3(0.99, 0.99, 0.99), 0.01);
        material *glass = new dielectric(1.5);
        material *light = new diffuse_light(new constant_texture(vec3(60, 60, 60)));
        material *light_red = new diffuse_light(new constant_texture(vec3(70, 0, 0)));
        material *light_green = new diffuse_light(new constant_texture(vec3(0, 70, 0)));
        material *light_blue = new diffuse_light(new constant_texture(vec3(0, 0, 70)));

        // 纹理贴图
        material *image_sky_tex_front = new diffuse_light(new image_texture(2048, 2048, 4, image_texture::TextureCategory::SKYBOX_FRONT));
        material *image_sky_tex_back = new diffuse_light(new image_texture(2048, 2048, 4, image_texture::TextureCategory::SKYBOX_BACK));
        material *image_sky_tex_left = new diffuse_light(new image_texture(2048, 2048, 4, image_texture::TextureCategory::SKYBOX_LEFT));
        material *image_sky_tex_right = new diffuse_light(new image_texture(2048, 2048, 4, image_texture::TextureCategory::SKYBOX_RIGHT));
        material *image_sky_tex_up = new diffuse_light(new image_texture(2048, 2048, 4, image_texture::TextureCategory::SKYBOX_UP));
        material *image_sky_tex_down = new diffuse_light(new image_texture(2048, 2048, 4, image_texture::TextureCategory::SKYBOX_DOWN));

        // 如果没有这些语句，将会出现很大问题，后面的世界可以生成，但不能正确运行
        // 将以下的关于纹理贴图的顶点创建注释掉，你将可以复现这个问题
        vertex v1_statue(vec3(0.5, 2.0, 0.1), vec3(0, 0, 0), vec3(0, 0, 0), vec3(0, 0, 0));
        vertex v2_statue(vec3(0.5, 0.1, 0.1), vec3(0, 0, 0), vec3(0, 0, 0), vec3(1, 0, 0));
        vertex v3_statue(vec3(2.5, 0.1, 0.0), vec3(0, 0, 0), vec3(0, 0, 0), vec3(1, 1, 0));
        vertex v4_statue(vec3(2.5, 2.0, 0.0), vec3(0, 0, 0), vec3(0, 0, 0), vec3(0, 1, 0));

        vertex v1_ring(vec3(0.1, 2.0, 0.5), vec3(0, 0, 0), vec3(0, 0, 0), vec3(0, 0, 0));
        vertex v2_ring(vec3(0.1, 0.1, 0.5), vec3(0, 0, 0), vec3(0, 0, 0), vec3(1, 0, 0));
        vertex v3_ring(vec3(0.1, 0.1, 2.5), vec3(0, 0, 0), vec3(0, 0, 0), vec3(1, 1, 0));
        vertex v4_ring(vec3(0.1, 2.0, 2.5), vec3(0, 0, 0), vec3(0, 0, 0), vec3(0, 1, 0));

        vertex v1_skybox(vec3(0.1, 2.0, 0.5), vec3(0, 0, 0), vec3(0, 0, 0), vec3(0, 0, 0));

        vertex *skybox_vert_list;
        uint32_t *skybox_ind_list;
        gen_skybox_vertex_list(&skybox_vert_list, &skybox_ind_list, 200);
        printf("texture Imported done\n");

        int obj_index = 0;

        // list[obj_index++] = new sphere(vec3(0, -5, 0), 10, noise); // test skybox

        list[obj_index++] = new sphere(vec3(0, -1000.5, 0), 1000, mental_ground); // ground

        // list[obj_index++] = new sphere(vec3(0, 0.5, 0), 0.25, noise);
        // list[obj_index++] = new sphere(vec3(0.75, 0.5, -0.75), 0.25, glass);
        // list[obj_index++] = new sphere(vec3(-0.75, 0.5, 0.75), 0.25, mental_steel);

        //  list[obj_index++] = new sphere(vec3(0, 2, 0), 2, noise);
        // list[obj_index++] = new sphere(vec3(2, 2, -4), 2, glass);
        // list[obj_index++] = new sphere(vec3(-2, 2, 6), 2, mental_steel);

        // list[obj_index++] = new sphere(vec3(0, 15, 0), 2, light);
        // list[obj_index++] = new sphere(vec3(10, 15, 10), 2, light);
        // list[obj_index++] = new sphere(vec3(10, 15, -10), 2, light);
        // list[obj_index++] = new sphere(vec3(-10, 15, -10), 2, light);
        // list[obj_index++] = new sphere(vec3(-10, 15, 10), 2, light);

        // list[obj_index++] = new sphere(vec3(0, 0, 0), 0.7, mental_steel); // zero point reference
        // list[obj_index++] = new triangle(v1_statue, v2_statue, v3_statue, image_statue_tex);
        // list[obj_index++] = new triangle(v1_statue, v3_statue, v4_statue, image_statue_tex);
        // list[obj_index++] = new triangle(v1_ring, v2_ring, v3_ring, image_ring_lord_tex);
        // list[obj_index++] = new triangle(v1_ring, v3_ring, v4_ring, image_ring_lord_tex);
        // list[obj_index++] = new sphere(vec3(0, 0, -1), 0.5, diffuse_steelblue);
        // list[obj_index++] = new sphere(vec3(1, 0, -1), 0.5, mental_copper);
        // list[obj_index++] = new sphere(vec3(-1, 0, -1), -0.45, glass);
        uint32_t sky_box_ind_list[] = {1, 0, 3, 2, 1, 3};

        list[obj_index++] = new models(skybox_vert_list + 0, sky_box_ind_list, 6, image_sky_tex_front, models::HitMethod::NAIVE, models::PrimType::TRIANGLE);
        list[obj_index++] = new models(skybox_vert_list + 4, sky_box_ind_list, 6, image_sky_tex_back, models::HitMethod::NAIVE, models::PrimType::TRIANGLE);
        list[obj_index++] = new models(skybox_vert_list + 8, sky_box_ind_list, 6, image_sky_tex_left, models::HitMethod::NAIVE, models::PrimType::TRIANGLE);
        list[obj_index++] = new models(skybox_vert_list + 12, sky_box_ind_list, 6, image_sky_tex_right, models::HitMethod::NAIVE, models::PrimType::TRIANGLE);
        list[obj_index++] = new models(skybox_vert_list + 16, sky_box_ind_list, 6, image_sky_tex_up, models::HitMethod::NAIVE, models::PrimType::TRIANGLE);
        list[obj_index++] = new models(skybox_vert_list + 20, sky_box_ind_list, 6, image_sky_tex_down, models::HitMethod::NAIVE, models::PrimType::TRIANGLE);
        // list[obj_index++] = new models(vertList, indList, 13500, mental_copper, models::HitMethod::NAIVE, models::PrimType::TRIANGLE);

        // printf("models count = %d\n", model_counts);
        // for (int models_index = 0; models_index < model_counts; models_index++)
        // {
        //     int model_ind_len = indOffset[models_index + 1] - indOffset[models_index + 0];
        //     printf("modelLen = %d\n", model_ind_len);
        //     list[obj_index++] = new models(&(vertList[vertOffset[models_index]]), &(indList[indOffset[models_index]]), model_ind_len, diffuse_steelblue, models::HitMethod::NAIVE, models::PrimType::TRIANGLE);
        // }
        int models_index = 0;
        // 无加速结构构造 Object
        // list[obj_index++] = new models(&(vertList[vertOffset[models_index]]), &(indList[indOffset[models_index]]), indOffset[models_index + 1] - indOffset[models_index + 0], mental_copper, models::HitMethod::NAIVE, models::PrimType::TRIANGLE);
        // BVH_Tree 加速结构
        list[obj_index++] = new models(&(vertList[vertOffset[models_index]]), &(indList[indOffset[models_index]]), indOffset[models_index + 1] - indOffset[models_index + 0], mental_copper, models::HitMethod::BVH_TREE, models::PrimType::TRIANGLE);
        models_index++;
        list[obj_index++] = new models(&(vertList[vertOffset[models_index]]), &(indList[indOffset[models_index]]), indOffset[models_index + 1] - indOffset[models_index + 0], glass, models::HitMethod::BVH_TREE, models::PrimType::TRIANGLE);
        models_index++;
        list[obj_index++] = new models(&(vertList[vertOffset[models_index]]), &(indList[indOffset[models_index]]), indOffset[models_index + 1] - indOffset[models_index + 0], noise, models::HitMethod::BVH_TREE, models::PrimType::TRIANGLE);

        *world = new hitable_list(list, obj_index);

        // printf("list[0] emission = %d\n", (*world)->objHasEmission());

        printf("world generate done, there are %d spearate obj in the world\n", (*world)->list_size);
    }
}
/* ##################################### 光线投射全局渲染 ##################################### */

__device__ ray get_ray_device(float s, float t, hiprandStateXORWOW *rand_state)
{
    vec3 temp01(1, 2, 3);
    vec3 temp02(3, 2, 1);

    temp02 = -temp01;

    // 全部相机参数
    vec3 u = PRIMARY_CAMERA.u;
    vec3 v = PRIMARY_CAMERA.v;
    float lens_radius = PRIMARY_CAMERA.lens_radius;
    float time0 = PRIMARY_CAMERA.time0, time1 = PRIMARY_CAMERA.time1;
    vec3 origin = PRIMARY_CAMERA.origin;
    vec3 upper_left_conner = PRIMARY_CAMERA.upper_left_conner;
    vec3 horizontal = PRIMARY_CAMERA.horizontal;
    vec3 vertical = PRIMARY_CAMERA.vertical;

    float hor_len = horizontal.length();
    float ver_len = vertical.length();

    vec3 rd = lens_radius * random_in_unit_disk_device(rand_state); // 得到设定光孔大小内的任意散点（即origin点——viewpoint）
    vec3 offset = rd.x() * u + rd.y() * v;                          // origin视点中心偏移（由xoy平面映射到u、v平面）
    offset = vec3(0, 0, 0);                                         // 这里目前有bug，先置为0
    float time = time0 + random_float_device(rand_state) * (time1 - time0);
    return ray(origin + offset, upper_left_conner + s * horizontal + t * vertical - origin - offset);

    // return ray();
    // return ray(origin, upper_left_conner + u * horizontal + v * vertical - origin);
}

__device__ vec3 shading_pixel(int depth, const ray &r, hitable_list **world, hiprandStateXORWOW *rand_state)
{

    // // 任务2023-04-09：着色函数改为直接光源采样 Render Equation is true

    // hit_record rec;

    // ray current_ray = r;
    // vec3 current_attenuation = vec3(1, 1, 1);
    // vec3 current_radiance = vec3(0, 0, 0);

    // // printf("emission test %d\n", world[0]->hasEmission());

    // for (int i = 0; i < depth; i++)
    // {

    //     // 如果与场景中的物体没有交点，则直接返回当前的 radiance
    //     if (!(*world)->hit(current_ray, 0.001, 999999, rec))
    //     {
    //         return current_radiance;
    //     }
    //     // 如果与场景中的物体有交点，且击中的位点处的材质发光（为一次/直接光源）
    //     if (rec.mat_ptr->hasEmission(0))
    //     {
    //         // 返回当前 radiance 值以及 当前亮度衰减系数与当前光源乘积的加和
    //         return current_radiance + current_attenuation * rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
    //     }

    //     // 第一步：开始一次/直接光源贡献值计算

    //     vec3 shade_point_coord = rec.p;
    //     vec3 shade_point_normal = rec.normal;
    //     shade_point_normal.make_unit_vector();
    //     double shade_point_distance = rec.t;

    //     vec3 L_dir(0, 0, 0);
    //     float light_pdf = 0.0;
    //     hit_record light_point;
    //     PRIMARY_CAMERA.sampleLight(light_point, light_pdf, world, rand_state);

    //     // printf("light_pdf = %f\n", light_pdf);

    //     vec3 light_point_coord = light_point.p;
    //     vec3 light_point_emit = light_point.mat_ptr->emitted(light_point.u, light_point.v, light_point.p);
    //     vec3 light_point_normal = light_point.normal;
    //     light_point_normal.make_unit_vector();

    //     double light_point_distance = (light_point_coord - shade_point_coord).length();

    //     vec3 shadePoint_to_viewPoint_wo = -current_ray.direction();
    //     vec3 directLightSource_to_shadePoint_wi = (shade_point_coord - light_point_coord);
    //     shadePoint_to_viewPoint_wo.make_unit_vector();
    //     directLightSource_to_shadePoint_wi.make_unit_vector();

    //     hit_record first_block_point;
    //     (*world)->hit(ray(shade_point_coord, -directLightSource_to_shadePoint_wi), 0.001, 999999, first_block_point);

    //     const float cos_theta_shadePoint = dot(shade_point_normal, -directLightSource_to_shadePoint_wi);
    //     const float cos_theta_lightPoint = dot(light_point_normal, directLightSource_to_shadePoint_wi);

    //     // 得到一次/直接光源的在当前位点的 BRDF
    //     vec3 BRDF_dir = rec.mat_ptr->computeBRDF(directLightSource_to_shadePoint_wi, shadePoint_to_viewPoint_wo, rec);
    //     // 得到一次/直接光源在当前位点的 其他衰减参数
    //     float parameter = cos_theta_lightPoint * cos_theta_shadePoint / pow(light_point_distance, 2) / light_pdf;
    //     // parameter = parameter < 0 ? -parameter : parameter;
    //     if (parameter < 0)
    //     {
    //         parameter = -parameter;
    //     }

    //     // printf("light_point_distance = %f\n", light_point_distance);
    //     // printf("parameter = %f\n", parameter);

    //     // 如果采样光源到当前位点的路径没有被其他物体遮挡
    //     if (first_block_point.t - light_point_distance > -0.005)
    //     {

    //         L_dir = light_point_emit * BRDF_dir * parameter;
    //         current_radiance += (L_dir * current_attenuation);
    //     }

    //     // 第二步：俄罗斯轮盘赌测试，随机断出
    //     if (PRIMARY_CAMERA.RussianRoulette < random_float_device(rand_state))
    //     {
    //         return current_radiance;
    //     }

    //     // 第三步：开始二次/间接光源贡献值计算

    //     vec3 BRDF_indir;
    //     vec3 L_indir(0, 0, 0);

    //     vec3 attenuation;
    //     ray scattered; // 获得散射光线，并将其更新到 current_ray

    //     // 如果未能成功散射，则直接返回当前 radiance
    //     // 这里将 current_ray 换成 r 你就能重现那个glass材质中间有一个亮环的奇怪问题/现象
    //     if (!rec.mat_ptr->scatter(current_ray, rec, attenuation, scattered, rand_state))
    //     {
    //         return current_radiance;
    //     }

    //     vec3 secondaryLightSource_to_shadePoint_wi = -scattered.direction();
    //     secondaryLightSource_to_shadePoint_wi.make_unit_vector();
    //     hit_record no_emit_obj;
    //     bool hitted = (*world)->hit(scattered, 0.0001, 999999, no_emit_obj);
    //     float cos_para;
    //     float para_indir;

    //     // 如果二次光线与场景中的物体有交点
    //     if (no_emit_obj.happened && hitted && no_emit_obj.t >= 0.005)
    //     {
    //         // 仅当 当前光线与场景交点的材质为lambertain，且散射射线与场景交点非光源时，直接返回当前亮度，不考虑之后的二次光线。
    //         // 这是由于对于金属镜面和透射表面，更多的光强是来源于直接散射带来的方向性较强的高光项，而非直接光源采样
    //         // 于是，即使有重复计算的成分，但总体影响不大
    //         if (no_emit_obj.mat_ptr->getMaterialType() == material::SelfMaterialType::LAMBERTAIN && no_emit_obj.mat_ptr->hasEmission(0))
    //         {
    //             return current_radiance;
    //         }
    //         else
    //         {

    //             // 首先应该将 scattered 散射光线更新到 current_ray
    //             current_ray = scattered;

    //             const float global_pdf = rec.mat_ptr->pdf(-shadePoint_to_viewPoint_wo, -secondaryLightSource_to_shadePoint_wi, shade_point_normal);

    //             // printf("flobal pdf = %d\n", global_pdf);
    //             // 得到二次/间接光源的在当前位点的 BRDF
    //             BRDF_indir = rec.mat_ptr->computeBRDF(secondaryLightSource_to_shadePoint_wi, shadePoint_to_viewPoint_wo, rec);
    //             cos_para = dot(-secondaryLightSource_to_shadePoint_wi, shade_point_normal);

    //             // 对于折射光所必要考虑的一步
    //             cos_para = cos_para < 0 ? -cos_para : cos_para;
    //             // if (cos_para < 0)
    //             // {
    //             //     cos_para = -cos_para;
    //             // }

    //             // 得到一次/直接光源在当前位点的 其他衰减参数
    //             para_indir = cos_para / PRIMARY_CAMERA.RussianRoulette / global_pdf;

    //             // 这里不支持递归，要进行修改
    //             // L_indir = shading_pixel(depth - 1, scattered, world, rand_state) * BRDF_indir * para_indir;
    //             // L_indir = vec3(0, 0, 0);
    //             current_attenuation *= (BRDF_indir * para_indir * attenuation);

    //             // printf("current attenuation = [%f,%f,%f]", BRDF_indir.e[0], BRDF_indir.e[1], BRDF_indir.e[2]);
    //         }
    //     }
    // }

    // // 因 depth 过大而断出，直接返回其当前 radiance 😃
    // return current_radiance;

    hit_record rec;
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    vec3 cur_radiance = vec3(0, 0, 0);
    for (int i = 0; i < depth; i++)
    {
        if ((*world)->hit(cur_ray, 0.001f, 999999, rec))
        {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, rand_state))
            {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else if (rec.mat_ptr->hasEmission(0))
            {
                return cur_attenuation * rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            }
            else
            {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else
        {

            return cur_attenuation * vec3(0.1, 0.1, 0.1); // 默认环境光

            // vec3 unit_direction = unit_vector(cur_ray.direction());
            // float t = 0.5f * (unit_direction.y() + 1.0f);
            // vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            // return cur_attenuation * c;
        }
    }
    return cur_attenuation * vec3(0.1, 0.1, 0.1);
}

__global__ void cuda_shading_unit(vec3 *frame_buffer, hitable_list **world, hiprandStateXORWOW *rand_state)
{
    int row_index = blockDim.y * blockIdx.y + threadIdx.y; // 当前线程所在行索引
    int col_index = blockDim.x * blockIdx.x + threadIdx.x; // 当前线程所在列索引

    if ((row_index >= FRAME_HEIGHT) || (col_index >= FRAME_WIDTH))
    {
        return;
    }

    int row_len = FRAME_WIDTH; // 行宽（列数）
    // int col_len = FRAME_HEIGHT;                           // 列高（行数）
    int global_index = (row_len * row_index + col_index); // 全局索引
    hiprandStateXORWOW local_rand_state = rand_state[global_index];

    vec3 col(0, 0, 0);

    /**
     *  2023-04-11
     *  我们发现程序的效率不尽如人意，尽管在十分简单的场景下，做到实时（30+fps）也十分困难。
     *  现在，我们从这里出发，查看到底是哪里占用了太多的时间，导致程序的效率低下
     * */

    /**
     *  首先，我们注释掉以下的关键计算部分，停止向场景内投射射线，查看计算用时。
     *  如果只是取消掉向场景投射，取消像素值计算部分，则用时为0.19ms
     *  取消像素的归一化以及计算等操作，这个用时将减少到0.055ms
     * */
    // random_float_device(&local_rand_state);
    for (int s = 0; s < PRIMARY_CAMERA.spp; s++)
    {
        float u = float(col_index + random_float_device(&local_rand_state)) / float(FRAME_WIDTH);
        float v = float(row_index + random_float_device(&local_rand_state)) / float(FRAME_HEIGHT);

        ray kernal_ray = get_ray_device(u, v, &local_rand_state);
        col += shading_pixel(BOUNCE_DEPTH, kernal_ray, world, &local_rand_state);
    }
    rand_state[global_index] = local_rand_state;
    col /= float(PRIMARY_CAMERA.spp);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);

    col = color_unit_normalization(col, 1);
    frame_buffer[global_index] = col;
}

/* ##################################### main 函数入口 ##################################### */

__host__ void init_and_render(void)
{

    // printf("size of bvh node = %d\n", sizeof(bvh_node));
    // printf("size of aabb = %d\n", sizeof(aabb));
    // printf("size of vec3 = %d\n", sizeof(vec3));
    // printf("size of bvh node * = %d\n", sizeof(bvh_node *));
    // printf("size of triangle * = %d\n", sizeof(triangle *));
    // printf("size of int = %d\n", sizeof(int));
    // printf("size of float = %d\n", sizeof(float));

    int device = 0;        // 设置使用第0块GPU进行运算
    hipSetDevice(device); // 设置运算显卡
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device); // 获取对应设备属性

    unsigned int block_size_width = 16;
    unsigned int block_size_height = 16;
    unsigned int grid_size_width = FRAME_WIDTH / block_size_width + 1;
    unsigned int grid_size_height = FRAME_HEIGHT / block_size_height + 1;
    dim3 dimBlock(block_size_width, block_size_height);
    dim3 dimGrid(grid_size_width, grid_size_height);

    /* ##################################### 纹理导入01 ##################################### */
    import_tex();

    /* ################################### 模型文件导入01 ################################### */
    vertex *vertList_host;
    uint32_t *indList_host;
    int *vertex_offset_host;
    int *ind_offset_host;
    std::vector<std::string> models_paths_host;

    // models_paths_host.push_back("../Models/viking/viking_room.obj");
    models_paths_host.push_back("../Models/bunny/bunny_low_resolution.obj");
    models_paths_host.push_back("../Models/bunny/bunny_x.obj");
    models_paths_host.push_back("../Models/bunny/bunny_z.obj");

    import_obj_from_file(&vertList_host, &vertex_offset_host, &indList_host, &ind_offset_host, models_paths_host);

    size_t vert_len = vertex_offset_host[models_paths_host.size()];
    size_t ind_len = ind_offset_host[models_paths_host.size()];

    vertex *vertList_device;
    uint32_t *indList_device;
    int *vertex_offset_device;
    int *ind_offset_device;

    hipMalloc((void **)&vertList_device, vert_len * sizeof(vertex));
    hipMalloc((void **)&indList_device, ind_len * sizeof(uint32_t));
    hipMalloc((void **)&vertex_offset_device, (models_paths_host.size() + 1) * sizeof(int));
    hipMalloc((void **)&ind_offset_device, (models_paths_host.size() + 1) * sizeof(int));

    hipMemcpy(vertList_device, vertList_host, vert_len * sizeof(vertex), hipMemcpyHostToDevice);
    hipMemcpy(indList_device, indList_host, ind_len * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(vertex_offset_device, vertex_offset_host, (models_paths_host.size() + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ind_offset_device, ind_offset_host, (models_paths_host.size() + 1) * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    /* ##################################### 随机数初始化 ##################################### */
    hiprandStateXORWOW *states;
    hipMalloc((void **)&states, sizeof(hiprandStateXORWOW) * FRAME_WIDTH * FRAME_HEIGHT);
    initialize_device_random<<<dimGrid, dimBlock>>>(states, time(nullptr), FRAME_WIDTH * FRAME_HEIGHT);
    hipDeviceSynchronize();
    // hiprandStateXORWOW *states = init_rand(block_size_width, block_size_height);

    /* ##################################### 摄像机初始化 ##################################### */
    cameraCreateInfo primaryCamera{};
    // primaryCamera.lookfrom = vec3(3, 2, 4);
    // primaryCamera.lookfrom = vec3(0, 1, 3);
    primaryCamera.lookfrom = vec3(2.5, 1, 2.5);
    // primaryCamera.lookfrom = vec3(20, 15, 20);
    primaryCamera.lookat = vec3(0, 0, 0);
    // primaryCamera.lookat = vec3(0.5, 0, 0.5);
    primaryCamera.up_dir = vec3(0, 1, 0);
    primaryCamera.fov = 40;
    primaryCamera.aspect = float(FRAME_WIDTH) / float(FRAME_HEIGHT);
    primaryCamera.focus_dist = 10.0; // 这里是焦距
    primaryCamera.aperture = 1;
    primaryCamera.t0 = 0.0;
    primaryCamera.t1 = 1.0;
    primaryCamera.RussianRoulette = 0.8;
    primaryCamera.frame_width = FRAME_WIDTH;
    primaryCamera.frame_height = FRAME_HEIGHT;

    primaryCamera.spp = 10;
    camera *cpu_camera = new camera(primaryCamera);
    int camera_size = sizeof(camera);
    hipMemcpyToSymbol(HIP_SYMBOL(PRIMARY_CAMERA), cpu_camera, camera_size);
    hipDeviceSynchronize();
    // init_camera();

    /* ##################################### 场景初始化 ##################################### */
    // 现在将 world 直接明确指定为一个 hitablelist
    // hitable **world_device;
    hitable_list **world_device;
    hitable **list_device;
    hipMalloc((void **)&world_device, sizeof(hitable_list *)); // 只留一个指针接口～ 其中的obj在列表中创建
    hipMalloc((void **)&list_device, sizeof(hitable *));       //

    gen_world<<<1, 1>>>(states, world_device, list_device, vertList_device, indList_device, vertex_offset_device, ind_offset_device, models_paths_host.size());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    // hitable **world = init_world(states);
    hipDeviceSynchronize();

    /* ################################## 初始化 CUDA 计时器 ################################## */
    hipEvent_t start, stop;
    float time_cost = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* ##################################### 全局渲染入口 ##################################### */

    /**
     *  以下将渲染改为一个loop，在这个loop中，我们可以对渲染中的某些参数进行修改，从而使得在实时渲染过程中
     * 拥有一些可交互的效果，比如目前将要实现的相机参数修改，这使得我们可以在场景中进行“游历”。
     *
     * */

    // 初始化帧缓存
    vec3 *frame_buffer_device;
    int size = FRAME_WIDTH * FRAME_HEIGHT * sizeof(vec3);
    hipMalloc((void **)&frame_buffer_device, size);
    size_t loop_count = 0;
    // 主机开辟帧缓存
    vec3 *frame_buffer_host = new vec3[FRAME_WIDTH * FRAME_HEIGHT];
    while (++loop_count)
    {

        // 首先使用当前参数进行渲染当前帧
        hipEventRecord(start); // device端 开始计时
        // 真正占用时间的渲染口
        // cuda_shading_unit<<<dim3(64, 32), dim3(8, 8)>>>(frame_buffer_device, world_device, states);
        cuda_shading_unit<<<dimGrid, dimBlock>>>(frame_buffer_device, world_device, states);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("CUDA Error when running cuda shading_unit: %s\n", hipGetErrorString(err));
            // Possibly: exit(-1) if program cannot continue....
        }
        hipEventRecord(stop); // device端 计时结束
        hipDeviceSynchronize();
        hipEventSynchronize(stop); // 计时同步

        hipEventElapsedTime(&time_cost, start, stop); // 计算用时，单位为ms
        std::cout << "This is " << loop_count << " frame, current render loop cost = " << time_cost << "ms" << std::endl;

        // 数据拷贝 & 本地写文件
        hipMemcpy(frame_buffer_host, frame_buffer_device, size, hipMemcpyDeviceToHost);
        std::string path = "../PicFlow/frame" + std::to_string(loop_count) + ".ppm";
        write_file(path, frame_buffer_host);

        // // 数据拷贝 & 图片流输出
        // hipMemcpy(frame_buffer_host, frame_buffer_device, size, hipMemcpyDeviceToHost);
        // cv::namedWindow("Image Flow");
        // // 一直执行这个循环，并将图像给到OpenCV创建的 window，直到按下 Esc 键推出
        // showFrameFlow(FRAME_WIDTH, FRAME_HEIGHT, frame_buffer_host);

        // if (cv::waitKey(1) == 27)
        // {
        //     break;
        // }

        // // 在 host 端更改相机参数
        // cpu_camera = modifyCamera(primaryCamera, loop_count);
        // // 将更改好的相机参数传递给device端的常量内存
        // hipMemcpyToSymbol(HIP_SYMBOL(PRIMARY_CAMERA), cpu_camera, camera_size);
        // hipDeviceSynchronize();

        // 断出条件
        // 当仅渲染一帧做测试时只需要将其设为1即可
        if (loop_count >= 1)
        {
            loop_count = 0;
            break;
        }
    }

    hipFree(frame_buffer_device);
    hipFree(world_device);
    hipFree(list_device);
    hipFree(states);
    hipFree(vertList_device);
    hipFree(indList_device);
    hipFree(vertex_offset_device);
    hipFree(ind_offset_device);
}

__host__ static void write_file(std::string file_path, vec3 *frame_buffer)
{
    std::ofstream OutputImage;
    OutputImage.open(file_path);
    OutputImage << "P3\n"
                << FRAME_WIDTH << " " << FRAME_HEIGHT << "\n255\n";

    for (int row = 0; row < FRAME_HEIGHT; row++)
    {
        for (int col = 0; col < FRAME_WIDTH; col++)
        {
            const int global_index = row * FRAME_WIDTH + col;
            vec3 pixelVal = frame_buffer[global_index];
            int ir = int(255.99 * pixelVal[0]);
            if (ir < 0)
                ir = 0;
            int ig = int(255.99 * pixelVal[1]);
            if (ig < 0)
                ig = 255;
            int ib = int(255.99 * pixelVal[2]);
            if (ib < 0)
                ib = 0;
            OutputImage << ir << " " << ig << " " << ib << "\n";
        }
    }
}

__host__ static void showFrameFlow(int width, int height, vec3 *frame_buffer_host)
{

    cv::Mat img = cv::Mat(cv::Size(width, height), CV_8UC3);

    for (int row = 0; row < FRAME_HEIGHT; row++)
    {
        for (int col = 0; col < FRAME_WIDTH; col++)
        {
            const int global_index = row * FRAME_WIDTH + col;
            vec3 pixelVal = frame_buffer_host[global_index];
            int ir = int(255.99 * pixelVal[0]);
            if (ir < 0)
                ir = 0;
            int ig = int(255.99 * pixelVal[1]);
            if (ig < 0)
                ig = 255;
            int ib = int(255.99 * pixelVal[2]);
            if (ib < 0)
                ib = 0;

            img.at<unsigned char>(row, col * 3 + 0) = ib;
            img.at<unsigned char>(row, col * 3 + 1) = ig;
            img.at<unsigned char>(row, col * 3 + 2) = ir;
        }
    }

    cv::imshow("Image Flow", img);
}
