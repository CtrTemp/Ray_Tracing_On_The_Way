#include "hip/hip_runtime.h"
#include "render.h"

/* ##################################### 随机数初始化 ##################################### */

__global__ void initialize_device_random(hiprandStateXORWOW *states, unsigned long long seed, size_t size)
{
    int row_index = blockDim.y * blockIdx.y + threadIdx.y; // 当前线程所在行索引
    int col_index = blockDim.x * blockIdx.x + threadIdx.x; // 当前线程所在列索引
    if ((row_index >= FRAME_HEIGHT) || (col_index >= FRAME_WIDTH))
    {
        return;
    }
    int row_len = FRAME_WIDTH; // 行宽（列数）
    // int col_len = FRAME_HEIGHT;                 // 列高（行数）
    int global_index = (row_len * row_index + col_index); // 全局索引

    hiprand_init(seed, global_index, 0, &states[global_index]);
}

/* ##################################### 摄像机初始化 ##################################### */

__host__ camera *createCamera(void)
{
    cameraCreateInfo createCamera{};
    // createCamera.lookfrom = vec3(-2, 2, 1);
    // createCamera.lookat = vec3(0, 0, -1);
    createCamera.lookfrom = vec3(2, 2, 2);
    createCamera.lookat = vec3(0, 0, 0);

    createCamera.up_dir = vec3(0, 1, 0);
    createCamera.fov = 40;
    createCamera.aspect = float(FRAME_WIDTH) / float(FRAME_HEIGHT);
    createCamera.focus_dist = 10.0; // 这里是焦距
    createCamera.aperture = 1;
    createCamera.t0 = 0.0;
    createCamera.t1 = 1.0;
    createCamera.frame_width = FRAME_WIDTH;
    createCamera.frame_height = FRAME_HEIGHT;

    createCamera.spp = 1;

    // 学会像vulkan那样构建
    return new camera(createCamera);
}

/* ##################################### 场景初始化 ##################################### */

__global__ void gen_world(hiprandStateXORWOW *rand_state, hitable **world, hitable **list)
{

    // 在设备端创建
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        material *noise = new lambertian(new noise_texture(2.5, rand_state));
        material *diffuse_steelblue = new lambertian(new constant_texture(vec3(0.1, 0.2, 0.5)));
        material *mental_copper = new mental(vec3(0.8, 0.6, 0.2), 0.1);
        material *glass = new dielectric(1.5);
        material *light = new diffuse_light(new constant_texture(vec3(6, 6, 6)));
        material *light_red = new diffuse_light(new constant_texture(vec3(70, 0, 0)));
        material *light_green = new diffuse_light(new constant_texture(vec3(0, 70, 0)));
        material *light_blue = new diffuse_light(new constant_texture(vec3(0, 0, 70)));

        vertex v1(vec3(1, 0, 0)), v2(vec3(0, 1, 0)), v3(vec3(0, 0, 1));
        triangle t1(v1, v2, v3, light_red);

        int obj_index = 0;

        list[obj_index++] = new sphere(vec3(0, -100.5, -1), 100, noise); // ground
        list[obj_index++] = new triangle(v1, v2, v3, diffuse_steelblue);
        // list[obj_index++] = new sphere(vec3(0, 0, -1), 0.5, diffuse_steelblue);
        // list[obj_index++] = new sphere(vec3(1, 0, -1), 0.5, mental_copper);
        // list[obj_index++] = new sphere(vec3(-1, 0, -1), -0.45, glass);
        *world = new hitable_list(list, 2);
    }
}
/* ##################################### 光线投射全局渲染 ##################################### */

__device__ ray get_ray_device(float s, float t, hiprandStateXORWOW *rand_state)
{
    vec3 temp01(1, 2, 3);
    vec3 temp02(3, 2, 1);

    temp02 = -temp01;
    
    // 全部相机参数
    vec3 u = PRIMARY_CAMERA.u;
    vec3 v = PRIMARY_CAMERA.v;
    float lens_radius = PRIMARY_CAMERA.lens_radius;
    float time0 = PRIMARY_CAMERA.time0, time1 = PRIMARY_CAMERA.time1;
    vec3 origin = PRIMARY_CAMERA.origin;
    vec3 upper_left_conner = PRIMARY_CAMERA.upper_left_conner;
    vec3 horizontal = PRIMARY_CAMERA.horizontal;
    vec3 vertical = PRIMARY_CAMERA.vertical;

    float hor_len = horizontal.length();
    float ver_len = vertical.length();

    vec3 rd = lens_radius * random_in_unit_disk_device(rand_state); // 得到设定光孔大小内的任意散点（即origin点——viewpoint）
    vec3 offset = rd.x() * u + rd.y() * v;                          // origin视点中心偏移（由xoy平面映射到u、v平面）
    offset = vec3(0, 0, 0);                                         // 这里目前有bug，先置为0
    float time = time0 + random_float_device(rand_state) * (time1 - time0);
    return ray(origin + offset, upper_left_conner + s * horizontal + t * vertical - origin - offset);
    // return ray(origin, upper_left_conner + u * horizontal + v * vertical - origin);
}

__device__ vec3 shading_pixel(int depth, const ray &r, hitable **world, hiprandStateXORWOW *rand_state)
{

    hit_record rec;
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < depth; i++)
    {
        if ((*world)->hit(cur_ray, 0.001f, 999999, rec))
        {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, rand_state))
            {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else if (rec.mat_ptr->hasEmission())
            {
                return rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            }
            else
            {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else
        {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.90, 0.0, 0.0);
}
__global__ void cuda_shading_unit(vec3 *frame_buffer, hitable **world, hiprandStateXORWOW *rand_state)
{
    int row_index = blockDim.y * blockIdx.y + threadIdx.y; // 当前线程所在行索引
    int col_index = blockDim.x * blockIdx.x + threadIdx.x; // 当前线程所在列索引

    if ((row_index >= FRAME_HEIGHT) || (col_index >= FRAME_WIDTH))
    {
        return;
    }

    int row_len = FRAME_WIDTH; // 行宽（列数）
    // int col_len = FRAME_HEIGHT;                           // 列高（行数）
    int global_index = (row_len * row_index + col_index); // 全局索引
    hiprandStateXORWOW local_rand_state = rand_state[global_index];

    vec3 col(0, 0, 0);
    // random_float_device(&local_rand_state)
    for (int s = 0; s < PRIMARY_CAMERA.spp; s++)
    {
        float u = float(col_index + random_float_device(&local_rand_state)) / float(FRAME_WIDTH);
        float v = float(row_index + random_float_device(&local_rand_state)) / float(FRAME_HEIGHT);

        ray kernal_ray = get_ray_device(u, v, &local_rand_state);
        col += shading_pixel(BOUNCE_DEPTH, kernal_ray, world, &local_rand_state);
    }
    rand_state[global_index] = local_rand_state;
    col /= float(PRIMARY_CAMERA.spp);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);

    frame_buffer[global_index] = col;
}

/* ##################################### main 函数入口 ##################################### */

__host__ void init_and_render(void)
{
    int device = 0;        // 设置使用第0块GPU进行运算
    hipSetDevice(device); // 设置运算显卡
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device); // 获取对应设备属性

    unsigned int block_size_width = 32;
    unsigned int block_size_height = 32;
    unsigned int grid_size_width = FRAME_WIDTH / block_size_width + 1;
    unsigned int grid_size_height = FRAME_HEIGHT / block_size_height + 1;
    dim3 dimBlock(block_size_width, block_size_height);
    dim3 dimGrid(grid_size_width, grid_size_height);

    /* ##################################### 随机数初始化 ##################################### */
    hiprandStateXORWOW *states;
    hipMalloc((void **)&states, sizeof(hiprandStateXORWOW) * FRAME_WIDTH * FRAME_HEIGHT);
    initialize_device_random<<<dimGrid, dimBlock>>>(states, time(nullptr), FRAME_WIDTH * FRAME_HEIGHT);
    hipDeviceSynchronize();
    // hiprandStateXORWOW *states = init_rand(block_size_width, block_size_height);

    /* ##################################### 摄像机初始化 ##################################### */
    int camera_size = sizeof(camera);
    camera *cpu_camera = createCamera();
    hipMemcpyToSymbol(HIP_SYMBOL(PRIMARY_CAMERA), cpu_camera, camera_size);
    hipDeviceSynchronize();
    // init_camera();

    /* ##################################### 场景初始化 ##################################### */
    hitable **world_device;
    hitable **list_device;
    hipMalloc((void **)&world_device, 15 * sizeof(hitable *));
    hipMalloc((void **)&list_device, sizeof(hitable *));
    gen_world<<<1, 1>>>(states, world_device, list_device);
    // hitable **world = init_world(states);

    /* ################################## 初始化 CUDA 计时器 ################################## */
    hipEvent_t start, stop;
    float time_cost = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* ##################################### 全局渲染入口 ##################################### */
    // 初始化帧缓存
    vec3 *frame_buffer_device;
    int size = FRAME_WIDTH * FRAME_HEIGHT * sizeof(vec3);
    hipMalloc((void **)&frame_buffer_device, size);
    hipEventRecord(start); // device端 开始计时
    cuda_shading_unit<<<dimGrid, dimBlock>>>(frame_buffer_device, world_device, states);
    hipEventRecord(stop); // device端 计时结束
    hipDeviceSynchronize();
    hipEventSynchronize(stop); // 计时同步

    hipEventElapsedTime(&time_cost, start, stop); // 计算用时，单位为ms
    // 停止计时
    std::cout << ": The total time of the pirmary loop is: " << time_cost << "ms" << std::endl;

    /* #################################### host端写图像文件 #################################### */

    // 在主机开辟 framebuffer 空间
    vec3 *frame_buffer_host = new vec3[FRAME_WIDTH * FRAME_HEIGHT];
    hipMemcpy(frame_buffer_host, frame_buffer_device, size, hipMemcpyDeviceToHost);
    // vec3 *frame_buffer_host = new vec3[FRAME_WIDTH * FRAME_HEIGHT];
    // render(block_size_height, block_size_height, states, world, frame_buffer_host);

    std::string file_path = "./any.ppm";
    std::ofstream OutputImage;
    OutputImage.open(file_path);
    OutputImage << "P3\n"
                << FRAME_WIDTH << " " << FRAME_HEIGHT << "\n255\n";

    for (int row = 0; row < FRAME_HEIGHT; row++)
    {
        for (int col = 0; col < FRAME_WIDTH; col++)
        {
            const int global_index = row * FRAME_WIDTH + col;
            vec3 pixelVal = frame_buffer_host[global_index];
            int ir = int(255.99 * pixelVal[0]);
            int ig = int(255.99 * pixelVal[1]);
            int ib = int(255.99 * pixelVal[2]);
            OutputImage << ir << " " << ig << " " << ib << "\n";
        }
    }

    std::cout << "Render Loop ALL DONE" << std::endl;
}
