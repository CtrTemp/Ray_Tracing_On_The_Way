#include "hip/hip_runtime.h"

/**
 *  我们应该考虑如何为C++的类内函数进行加速。
 *  首先一点就是你绝对不能将一个C++类文件直接改写成.cu文件，因为类内函数无法内联成C编译输出，这样在Cpp文件中
 * 调用就会存在未定义的错误。并且换个方向来讲，类内函数会有大量的函数重载，即便可以使用 extern "C" 强制使其
 * 转换成C编译成的函数名称，也会存在大量的函数重定义现象。所以无论如何你不能这样做。
 *  那么如果想使用CUDA加速类内函数，目前的唯一做法就是将具体需要加速的类内函数重新封装一次。具体做法就是原本的
 * 类的定义以及类内函数的实现还是分别使用.cpp以及.h文件实现；对于需要使用CUDA进行加速的类内函数，其具体实现我们
 * 单独放在另一个.cu文件中，并且使用.cuh文件内联成C编译输出，再将这个封装好的函数通过.cuh引入我们的类内函数。
 *  目前对于当前我们的camera.cpp文件，我们对于光线投射的部分用到CUDA加速，那么这部分我们就单独使用这个.cu文件
 * 重新实现一下那个对应的函数即可。
 * */

#include "cast_ray.cuh"

/**
 *  现在要思考的一个主要问题：要把那些变量设置在device端让GPU可见，那些设置在host端让CPU可见
 *  主要涉及：场景建立/相机摆放
 *  我的初步思考：这些都应该在初始化的时候静态建立在device端，然后在需要修改的时候host端发送指令进行修改。
 * 比如涉及场景中物体的移动或者相机的在场景中的漫游。
 *  这就使得你必须要在host端保留一份和device端一模一样的副本，用于进行修改维护，并更新device端的变量。
 * 所以现在看来最好的方法就是直接在host端建立，并拷贝到device端。
 * */

extern __constant__ camera PRIMARY_CAMERA;


__device__ ray get_ray_cu(float s, float t)
{
    vec3 rd = PRIMARY_CAMERA.lens_radius * random_in_unit_disk(); // 得到设定光孔大小内的任意散点（即origin点——viewpoint）
    // （该乘积的后一项是单位光孔）
    vec3 offset = rd.x() * PRIMARY_CAMERA.u + rd.y() * PRIMARY_CAMERA.v; // origin视点中心偏移（由xoy平面映射到u、v平面）
    // return ray(origin + offset, lower_left_conner + s*horizontal + t*vertical - origin - offset);
    float time = PRIMARY_CAMERA.time0 + drand48() * (PRIMARY_CAMERA.time1 - PRIMARY_CAMERA.time0);
    return ray(PRIMARY_CAMERA.origin + offset, PRIMARY_CAMERA.upper_left_conner + s * PRIMARY_CAMERA.horizontal + t * PRIMARY_CAMERA.vertical - PRIMARY_CAMERA.origin - offset, time);
}

__global__ void cuda_shading_unit(vec3 *frame_buffer)
{
    // 这里使用二维线程开辟

    int row_index = blockDim.y * blockIdx.y + threadIdx.y; // 当前线程所在行索引
    int col_index = blockDim.x * blockIdx.x + threadIdx.x; // 当前线程所在列索引

    int row_len = gridDim.x * blockDim.x;                 // 行宽（列数）
    int col_len = gridDim.y * blockDim.y;                 // 列高（行数）
    int global_index = (row_len * row_index + col_index); // 全局索引

    int global_size = row_len * col_len;
    float single_color = (float)(global_index) / global_size;
    vec3 color(0.9, 0.1, 0.8);
    // color[0] = single_color;
    // color[1] = single_color;
    // color[2] = single_color;

    // ray =

    frame_buffer[global_index] = color;
}

vec3 *cast_ray_cu(float frame_width, float frame_height, int spp)
{
    int device = 0;        // 设置使用第0块GPU进行运算
    hipSetDevice(device); // 设置运算显卡
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device); // 获取对应设备属性

    // 整个frame的大小
    // int size = frame_width * frame_height * sizeof(float);
    int size = frame_width * frame_height * sizeof(vec3);
    // 开辟将要接收计算回传数据的内存空间
    vec3 *frame_buffer_host = (vec3 *)malloc(size);

    unsigned int block_size_width = 32;
    unsigned int block_size_height = 32;
    unsigned int grid_size_width = frame_width / block_size_width;
    unsigned int grid_size_height = frame_height / block_size_height;

    // std::cout << "grid size = [" << grid_size_height << ", " << grid_size_width << "]" << std::endl;
    // std::cout << "global size = " << size << std::endl;

    dim3 dimBlock(block_size_height, block_size_width, 1);
    dim3 dimGrid(grid_size_height, grid_size_width, 1);

    // 开辟显存空间
    vec3 *frame_buffer_device;
    hipMalloc((void **)&frame_buffer_device, size);

    // ##################### 这里看一下并行用时 #####################

    hipEvent_t start, stop;
    float runTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // 不要忘了给模板函数添加模板参数
    // 所有的并行计算应该都在这一个函数中完成，这个函数要调用其他.cu文件中的函数，并且也要在device上执行
    // 关键问题是那些预定义的类怎么办？CUDA中无法直接使用这些类
    cuda_shading_unit<<<dimGrid, dimBlock>>>(frame_buffer_device);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);

    std::cout << ": para time cost: " << runTime << "ms" << std::endl;

    // ##################### End #####################

    // 从显存向内存拷贝（第一个参数是dst，第二个参数是src）
    hipMemcpy(frame_buffer_host, frame_buffer_device, size, hipMemcpyDeviceToHost);

    // std::cout << "host[1000] = " << frame_buffer_host[1000] << std::endl;

    // 你不能这样直接访问device的地址！！！
    // std::cout << "device = " << frame_buffer_device[0] << std::endl;

    hipFree(frame_buffer_device);

    return frame_buffer_host;
}
