#include "hip/hip_runtime.h"

/**
 *  我们应该考虑如何为C++的类内函数进行加速。
 *  首先一点就是你绝对不能将一个C++类文件直接改写成.cu文件，因为类内函数无法内联成C编译输出，这样在Cpp文件中
 * 调用就会存在未定义的错误。并且换个方向来讲，类内函数会有大量的函数重载，即便可以使用 extern "C" 强制使其
 * 转换成C编译成的函数名称，也会存在大量的函数重定义现象。所以无论如何你不能这样做。
 *  那么如果想使用CUDA加速类内函数，目前的唯一做法就是将具体需要加速的类内函数重新封装一次。具体做法就是原本的
 * 类的定义以及类内函数的实现还是分别使用.cpp以及.h文件实现；对于需要使用CUDA进行加速的类内函数，其具体实现我们
 * 单独放在另一个.cu文件中，并且使用.cuh文件内联成C编译输出，再将这个封装好的函数通过.cuh引入我们的类内函数。
 *  目前对于当前我们的camera.cpp文件，我们对于光线投射的部分用到CUDA加速，那么这部分我们就单独使用这个.cu文件
 * 重新实现一下那个对应的函数即可。
 * */

#include "cast_ray.cuh"

/**
 *  现在要思考的一个主要问题：要把那些变量设置在device端让GPU可见，那些设置在host端让CPU可见
 *  主要涉及：场景建立/相机摆放
 *  我的初步思考：这些都应该在初始化的时候静态建立在device端，然后在需要修改的时候host端发送指令进行修改。
 * 比如涉及场景中物体的移动或者相机的在场景中的漫游。
 *  这就使得你必须要在host端保留一份和device端一模一样的副本，用于进行修改维护，并更新device端的变量。
 * 所以现在看来最好的方法就是直接在host端建立，并拷贝到device端。
 * */

// extern __constant__ camera PRIMARY_CAMERA;

__device__ ray get_ray_cu(float s, float t, hiprandStateXORWOW *rand_state)
{

    // 全部相机参数
    vec3 u(0.707, 0, 0.707);
    vec3 v(0.3313, -0.8835, 0.3313);
    float lens_radius = 0.5;
    float time0 = 0, time1 = 1.0;
    vec3 origin(20, 15, 20);
    vec3 upper_left_conner(9.97, 13.53, 15.12);
    vec3 horizontal(5.15, 0, 5.15);
    vec3 vertical(2.41, -6.43, 2.41);

    // return ray();

    vec3 rd = lens_radius * random_in_unit_disk_device(rand_state); // 得到设定光孔大小内的任意散点（即origin点——viewpoint）
    // （该乘积的后一项是单位光孔）
    vec3 offset = rd.x() * u + rd.y() * v; // origin视点中心偏移（由xoy平面映射到u、v平面）
    // return ray(origin + offset, lower_left_conner + s*horizontal + t*vertical - origin - offset);
    float time = time0 + random_double_device(rand_state) * (time1 - time0);
    return ray(origin + offset, upper_left_conner + s * horizontal + t * vertical - origin - offset, time);
}

// 这个函数应该没有问题，出问题的应该是 ray 的获取函数，get_ray_cu() 出了问题
__device__ vec3 shading_cu(ray r)
{
    vec3 unit_direction = unit_vector(r.direction());
    auto t = 0.5 * (unit_direction.y() + 1.0);
    // return vec3(0.5, 0, 0);
    return (1.0 - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
}

__global__ void cuda_shading_unit(vec3 *frame_buffer, hiprandStateXORWOW_t *rand_state)
{

    /*################################ 全局索引 ################################*/
    int row_index = blockDim.y * blockIdx.y + threadIdx.y; // 当前线程所在行索引
    int col_index = blockDim.x * blockIdx.x + threadIdx.x; // 当前线程所在列索引

    int row_len = gridDim.x * blockDim.x;                 // 行宽（列数）
    int col_len = gridDim.y * blockDim.y;                 // 列高（行数）
    int global_index = (row_len * row_index + col_index); // 全局索引

    // int global_size = row_len * col_len;

    // /*############################## 随机数初始化 ##############################*/
    // hiprandStateXORWOW_t *rand_state;
    // hiprand_init(global_index, 0, 0, rand_state);

    // /*############################## 获取当前光线 ##############################*/
    // 原来是这里出了大问题！！最后一项访问不到
    float u = float(col_index + random_double_device(0, 1.0, &rand_state[global_index])) / float(512);
    float v = float(row_index + random_double_device(0, 1.0, &rand_state[global_index])) / float(512);
    ray kernal_ray = get_ray_cu(u, v, &rand_state[global_index]);
    vec3 color = shading_cu(kernal_ray);

    // float single_color = (float)(global_index) / global_size;
    // vec3 color(random_double_device(&rand_state[global_index]), random_double_device(&rand_state[global_index]), random_double_device(&rand_state[global_index]));
    // color[0] = single_color;
    // color[1] = single_color;
    // color[2] = single_color;

    frame_buffer[global_index] = color;
}

__global__ void initialize_device_random(hiprandStateXORWOW_t *states, unsigned long long seed, size_t size)
{
    /*################################ 全局索引 ################################*/
    int row_index = blockDim.y * blockIdx.y + threadIdx.y; // 当前线程所在行索引
    int col_index = blockDim.x * blockIdx.x + threadIdx.x; // 当前线程所在列索引

    int row_len = gridDim.x * blockDim.x;                 // 行宽（列数）
    int col_len = gridDim.y * blockDim.y;                 // 列高（行数）
    int global_index = (row_len * row_index + col_index); // 全局索引

    hiprand_init(seed, global_index, 0, &states[global_index]);
}

vec3 *cast_ray_cu(float frame_width, float frame_height, int spp)
{
    int device = 0;        // 设置使用第0块GPU进行运算
    hipSetDevice(device); // 设置运算显卡
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device); // 获取对应设备属性

    // 整个frame的大小
    // int size = frame_width * frame_height * sizeof(float);
    int size = frame_width * frame_height * sizeof(vec3);
    // 开辟将要接收计算回传数据的内存空间
    vec3 *frame_buffer_host = (vec3 *)malloc(size);

    unsigned int block_size_width = 32;
    unsigned int block_size_height = 32;
    unsigned int grid_size_width = frame_width / block_size_width;
    unsigned int grid_size_height = frame_height / block_size_height;

    // std::cout << "grid size = [" << grid_size_height << ", " << grid_size_width << "]" << std::endl;
    // std::cout << "global size = " << size << std::endl;

    dim3 dimBlock(block_size_height, block_size_width, 1);
    dim3 dimGrid(grid_size_height, grid_size_width, 1);

    /* ############################### 初始化随机数 ############################### */
    hiprandStateXORWOW_t *states;
    hipMalloc(&states, sizeof(hiprandStateXORWOW_t) * FRAME_WIDTH * FRAME_HEIGHT);

    initialize_device_random<<<dimGrid, dimBlock>>>(states, time(nullptr), frame_width * frame_height);

    hipDeviceSynchronize();

    /* ############################### Real Render ############################### */

    // 开辟显存空间
    vec3 *frame_buffer_device;
    hipMalloc((void **)&frame_buffer_device, size);

    // ##################### 这里看一下并行用时 #####################

    hipEvent_t start, stop;
    float runTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // 不要忘了给模板函数添加模板参数
    // 所有的并行计算应该都在这一个函数中完成，这个函数要调用其他.cu文件中的函数，并且也要在device上执行
    // 关键问题是那些预定义的类怎么办？CUDA中无法直接使用这些类
    cuda_shading_unit<<<dimGrid, dimBlock>>>(frame_buffer_device, states);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);

    std::cout << ": para time cost: " << runTime << "ms" << std::endl;

    // ##################### End #####################

    // 从显存向内存拷贝（第一个参数是dst，第二个参数是src）
    hipMemcpy(frame_buffer_host, frame_buffer_device, size, hipMemcpyDeviceToHost);

    // std::cout << "host[1000] = " << frame_buffer_host[1000] << std::endl;

    // 你不能这样直接访问device的地址！！！
    // std::cout << "device = " << frame_buffer_device[0] << std::endl;

    hipFree(frame_buffer_device);

    return frame_buffer_host;
}

void camera_initialization(void)
{
    int device = 0;        // 设置使用第0块GPU进行运算
    hipSetDevice(device); // 设置运算显卡
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device); // 获取对应设备属性

    /* ############################### 初始化摄像机 ############################### */
    int camera_size = sizeof(camera);

    // std::cout << "camera size = " << camera_size << std::endl;
    camera *cpu_camera = createCamera();

    // 将host本地创建初始化好的摄像机，连带参数一同拷贝到device设备端
    hipMemcpyToSymbol(HIP_SYMBOL(PRIMARY_CAMERA), cpu_camera, camera_size);

    hipDeviceSynchronize();

    // std::cout << "camera height = " << get_camera_info()->frame_height << std::endl;
}



